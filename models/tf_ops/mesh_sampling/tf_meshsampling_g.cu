
#include <hip/hip_runtime.h>
//Mesh Sampling Operator CuDA
//Author: Weiyue Wang
//Reference: https://github.com/charlesq34/pointnet-autoencoder/blob/master/tf_ops/nn_distance/tf_nndistance_g.cu
//           https://github.com/PointCloudLibrary/pcl/blob/master/tools/mesh_sampling.cpp

#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include <stdio.h>
#include <assert.h>

__device__ float TriangleArea(float *a, float *b, float *c){
    float side1 = 10 * sqrt ( (a[0]-b[0])*(a[0]-b[0]) + (a[1]-b[1])*(a[1]-b[1]) + (a[2]-b[2])*(a[2]-b[2]) );
    float side2 = 10 * sqrt ( (a[0]-c[0])*(a[0]-c[0]) + (a[1]-c[1])*(a[1]-c[1]) + (a[2]-c[2])*(a[2]-c[2]) );
    float side3 = 10 * sqrt ( (c[0]-b[0])*(c[0]-b[0]) + (c[1]-b[1])*(c[1]-b[1]) + (c[2]-b[2])*(c[2]-b[2]) );
    float s = (side1 + side2 + side3)/2;
    float area =  sqrt( s * (s - side1) * (s - side2) * (s - side3));

    return area;
}

__device__ void getPoint(const float *vertices, int v_id, float *p){
    p[0] = vertices[3* v_id];
    p[1] = vertices[3* v_id+1];
    p[2] = vertices[3* v_id+2];
}

__device__ void getFeat(const float *feats, int v_id, int n_c, float *p){

    for (int i = 0; i < n_c; i++)
        p[i] = feats[v_id + i];
}

__device__ void getTriangle(const int *triangles, int t_id, int &v1, int &v2, int &v3){
    v1 = triangles[3 * t_id];
    v2 = triangles[3 * t_id + 1];
    v3 = triangles[3 * t_id + 2];
}

__device__  int lower_bound (const float * array, int n, const float& val)
{
    int it, first=0;
    int step;
    int count = n-1;
    while (count>0)
    {
        it = first; step=count/2; it += step;
        if (array[it]<val) {
            first=++it;
            count-=step+1;
        }
        else{
            count=step;
        }
    }
    return first;
}

__device__ void randomPointTriangle_array (const float * A, const float * B, const float * C, const float r1, const float r2, float * p, int n_c){
    float r1sqr = std::sqrt (max(0.f, r1));
    float OneMinR1Sqr = (1 - r1sqr);
    float OneMinR2 = (1 - r2);
    for (int i = 0; i < n_c; i++){
        p[i] = r1sqr * r2 * C[i] + r1sqr * OneMinR2 * B[i] + OneMinR1Sqr * A[i];
    }
}

__device__  int randPSurface (const int *triangles, const float *vertices, const float * feats, const float * cumulativeAreas, int n_triangles, const float totalArea, const int n_c, float *p, float * outfeats, const float r, const float r1, const float r2){

    int el = (lower_bound(cumulativeAreas, n_triangles, r * totalArea));
    int v1, v2, v3;

    getTriangle(triangles, el, v1, v2, v3);

    randomPointTriangle_array (&vertices[3*v1], &vertices[3*v2], &vertices[3*v3], r1, r2, p, 3);
    randomPointTriangle_array (&feats[n_c*v1], &feats[n_c*v2], &feats[n_c*v3], r1, r2, outfeats, n_c);

    return el;
}

__global__ void MeshSamplingKernel(const int b, const int * nverts, const int maxnverts, const float * vertices, const int * ntriangles, const int maxntriangles, const int * triangles, const int n_c, const float * feats, const float * r, const float * r1, const float * r2, const int n_samples, const float * cumulativeAreas, float * points, float* outfeats, int * correspondingface){

    for (int i=blockIdx.x; i<b; i+=gridDim.x){
        int n_triangles = ntriangles[i];
        for (int sample_id=threadIdx.x+blockIdx.y*blockDim.x; sample_id < n_samples; sample_id+=blockDim.x*gridDim.y){
            correspondingface[(i*n_samples+sample_id)] = randPSurface (&triangles[i*maxntriangles*3], &vertices[i*maxnverts*3], &feats[i*maxnverts*n_c],
                                                                       &cumulativeAreas[i*maxntriangles], n_triangles, cumulativeAreas[i*maxntriangles+n_triangles-1], n_c,
                                                                       &points[(i*n_samples+sample_id)*3], &outfeats[(i*n_samples+sample_id)*n_c], r[(i*n_samples+sample_id)], r1[(i*n_samples+sample_id)], r2[(i*n_samples+sample_id)]);

        }
        __syncthreads();
    }
}

__global__ void CumulativeAreaKernel(const int b, const int * nverts, const int maxnverts, const float * vertices, const int * ntriangles, const int maxntriangles, const int * triangles, float * cumulativeAreas){

    for (int i=blockIdx.x; i<b; i+=gridDim.x){
        int n_triangles = ntriangles[i];
        int n_verts = nverts[i];

        assert (n_triangles <= maxntriangles);
        assert (n_verts <= maxnverts);

        float p1[3], p2[3], p3[3], totalArea = 0;
        int v1,v2,v3;
        for (int triangle_id=0; triangle_id < n_triangles; triangle_id++){
            getTriangle(&triangles[i*maxntriangles*3], triangle_id, v1, v2, v3);
            getPoint(&vertices[i*maxnverts*3], v1, p1);
            getPoint(&vertices[i*maxnverts*3], v2, p2);
            getPoint(&vertices[i*maxnverts*3], v3, p3);

            float area = TriangleArea(p1, p2, p3);

            if (!(isnan(area)))
                totalArea += area;

            cumulativeAreas[i*maxntriangles+triangle_id] = totalArea;
        }
    }
}


void MeshSamplingKernelLauncher( \
    /*inputs*/  const int b, const int * n_verts, const int maxn_verts, const float * vertices, const int * n_triangles, const int maxn_triangles, const int * triangles, const int n_c, const float * feats, const  float * r, const float * r1, const float * r2, const int n_samples, \
    /*outputs*/ float * points, float* outfeats, int * correspondingface){

    float *cumulativeAreas;
    cudaMalloc((void**)&cumulativeAreas, b*maxn_triangles*sizeof(float));

    CumulativeAreaKernel<<<64,1>>>(b, n_verts, maxn_verts, vertices, n_triangles, maxn_triangles, triangles,cumulativeAreas);
    MeshSamplingKernel<<<dim3(32,16,1),512>>>(b, n_verts, maxn_verts, vertices, n_triangles, maxn_triangles, triangles, n_c, feats, r, r1, r2, n_samples, cumulativeAreas, points, outfeats, correspondingface);
    cudaFree(cumulativeAreas);
}

/****************** Gradient ******************/
__device__ void gradrandomPointTriangle_array (float * A, float * B, float * C,
                                                int * count_A, int * count_B, int * count_C,
                                                const float r1, const float r2, const float * gp, const int n_c){
    float r1sqr = std::sqrt (max(0.f, r1));
    float OneMinR1Sqr = (1 - r1sqr);
    float OneMinR2 = (1 - r2);
    float oldA = A[0];
    for (int i = 0; i < n_c; i++){
        atomicAdd(&A[i], gp[i] *OneMinR1Sqr);//gp[i] * 
        atomicAdd(&B[i], gp[i] *r1sqr * OneMinR2);
        atomicAdd(&C[i], gp[i] *r1sqr * r2);
    }
    atomicAdd(count_A, 1);
    atomicAdd(count_B, 1);
    atomicAdd(count_C, 1);
}


__global__ void MeshSamplingGradKernel(const int b, const int maxnverts, const int maxntriangles, const int * triangles,
                                        const int n_c, const float * r1, const float * r2, const int n_samples, const float * grad_outfeat,
                                        const int * correspondingface, int* cumulativeCounts, float* grad_feats){
    for (int i=blockIdx.x;i<b;i+=gridDim.x){
        for (int sample_id=threadIdx.x+blockIdx.y*blockDim.x; sample_id<n_samples; sample_id+=blockDim.x*gridDim.y){
            // index outfeat: (i * n_samples+sample_id)*n_c
            // index infeat: (i * maxnverts+sample_id)*n_c
            int v1, v2, v3;

            getTriangle(&triangles[i*maxntriangles*3], correspondingface[(i*n_samples+sample_id)], v1, v2, v3);

            float * grad_feats_tmp = &grad_feats[i*maxnverts*n_c];
            int * cumulativeCounts_tmp = &cumulativeCounts[i*maxnverts];
            gradrandomPointTriangle_array(&grad_feats_tmp[n_c*v1], &grad_feats_tmp[n_c*v2], &grad_feats_tmp[n_c*v3],
                                          &cumulativeCounts_tmp[v1], &cumulativeCounts_tmp[v2], &cumulativeCounts_tmp[v3],
                                          r1[(i*n_samples+sample_id)], r2[(i*n_samples+sample_id)],
                                          &grad_outfeat[(i*n_samples+sample_id)*n_c], n_c);

        }
    }
}

__global__ void AvgGradKernel(const int b, const int maxnverts, const int n_c, float* grad_feats, int* cumulativeCounts){
    for (int i=blockIdx.x;i<b;i+=gridDim.x){
        for (int v_id=threadIdx.x+blockIdx.y*blockDim.x; v_id<maxnverts; v_id+=blockDim.x*gridDim.y){
            for (int i_c = 0; i_c < n_c; i_c++){
                if (cumulativeCounts[i*maxnverts+v_id]!=0){
                    grad_feats[i*maxnverts*n_c+v_id*n_c+i_c] /= (float)cumulativeCounts[i*maxnverts+v_id];
                }
            }
        }
    }
}

void MeshSamplingGradKernelLauncher(const int b, const int maxnverts, const int maxntriangles, const int * triangles,
                                    const int n_c, const float * r1, const float * r2, const int n_samples, const float * grad_outfeat,
                                    const int * correspondingface, float* grad_feats){
    int *cumulativeCounts;
    cudaMalloc((void**)&cumulativeCounts, b*maxnverts*sizeof(int));
    cudaMemset(grad_feats, 0, b*maxnverts*n_c*sizeof(float));
    cudaMemset(cumulativeCounts, 0, b*maxnverts*sizeof(int));

    MeshSamplingGradKernel<<<dim3(32,16,1),512>>>(b, maxnverts, maxntriangles, triangles, n_c, r1, r2, n_samples, grad_outfeat,
                                                correspondingface, cumulativeCounts, grad_feats);
    AvgGradKernel<<<dim3(32,16,1),512>>>(b, maxnverts, n_c, grad_feats, cumulativeCounts);
    cudaFree(cumulativeCounts);
}

#endif

